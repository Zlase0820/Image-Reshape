#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"  
#include ""  
#include <hip/hip_runtime.h>  
#include <cuda_device_runtime_api.h>  
#include <opencv2\gpu\gpu.hpp>  
#include <opencv2\gpu\gpumat.hpp>  
#include <opencv2\opencv.hpp>  
#include <opencv.hpp>  
#include <stdio.h>  
#include <iostream>  
#include "opencv2/gpu/device/common.hpp"  
#include "opencv2/gpu/device/reduce.hpp"  
#include "opencv2/gpu/device/functional.hpp"  
#include "opencv2/gpu/device/warp_shuffle.hpp"  
#include "windows.h"

#include "Binarize.h"

using namespace std;
using namespace cv;
using namespace gpu;

// �궨�� Block�ĳߴ�Ϊ 16*2
//#define DEF_BLOCK_X  16
//#define DEF_BLOCK_Y  2


// src_cpu_dataԭͼ��ָ�룻out_cpu_data�����ͼ��ָ�룻scale���䱶��
int NearestInterpolation(uchar* &src_cpu_data, uchar* &out_cpu_data, float scale,int rows,int cols,int channels,int out_rows,int out_cols)
{
	float f_src_row;
	int   i_src_row;

	float f_src_col;    //ԭͼ������float��
	int   i_src_col;    //ԭͼ������int��

	for (int y = 0; y < out_rows; y++)   
	{
		for (int x = 0; x < out_cols ; x++)
		{
			int a = 0, b = 0;
			f_src_row = y / scale;
			i_src_row = (int)f_src_row;
			
			f_src_col = x / scale;
			i_src_col = (int)f_src_col;			
			
			if ((f_src_row - i_src_row) >= 0.5 && i_src_row <(rows - 1) )    //i_out_row <(rows - 1)ֻ��Ϊ�˷�ֹ�����߿�
				a=1;

			if ((f_src_col - i_src_col) >= 0.5 && i_src_col < (cols - 1))
				b=1;

			*(out_cpu_data + 3 * x + y*out_cols*channels) = *(src_cpu_data + 3 * (i_src_col + a) + (i_src_row + b)*cols*channels);
			*(out_cpu_data + 3 * x + y*out_cols*channels+1) = *(src_cpu_data + 3 * (i_src_col + a) + (i_src_row + b)*cols*channels+1);
			*(out_cpu_data + 3 * x + y*out_cols*channels+2) = *(src_cpu_data + 3 * (i_src_col + a) + (i_src_row + b)*cols*channels+2);
		}
	}
	return 0;
}


template <int nthreads>
__global__ void NI_kernel(uchar* &src_gpu_data, uchar* &out_gpu_data, float scale, int rows, int cols, int channels, int out_rows, int out_cols)
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;  //x
	const int y = blockIdx.y * blockDim.y + threadIdx.y;  //y	

	printf("1");
	float f_src_col;
	int   i_src_col;
	float cha_col;

	float f_src_row;
	int   i_src_row;
	float cha_row;

	int a = 0, b = 0;

	f_src_col = x / scale;
	f_src_row = y / scale;
	i_src_col = (int)f_src_col;
	i_src_row = (int)f_src_row;
	cha_col   = f_src_col - i_src_col;
	cha_row   = f_src_row - i_src_row;

//-----------�°汾----------
//�ɰ汾ÿһ���߳���Ҫ��Ҫ�Ƚ�4�Σ��°汾ÿ���߳�ֻ��Ҫ�Ƚ�2�Ρ�
//�ɰ汾��ֵ����̫�������Ѽ�Ϊ���С�
//�ɰ汾ʹ����GpuMat�����汾ʹ��ָ����д��룬�����˺ںе���

	if (cha_col >= 0.5)  a++;
	if (cha_row >= 0.5)  b++;
	*(out_gpu_data + x + y*out_cols*channels) = *(src_gpu_data + (i_src_col + a) + (i_src_row + b)*cols*channels);
	


//	*(out_gpu_data + 3 * x + y*out_cols*channels) = *(src_gpu_data + 3 * (i_src_col + a) + (i_src_row + b)*cols*channels);
//	*(out_gpu_data + 3 * x + y*out_cols*channels + 1) = *(src_gpu_data + 3 * (i_src_col + a) + (i_src_row + b)*cols*channels + 1);
//	*(out_gpu_data + 3 * x + y*out_cols*channels + 2) = *(src_gpu_data + 3 * (i_src_col + a) + (i_src_row + b)*cols*channels + 2);
	
}



int main()
{
	float scale = 0.6f;   
	char* src_path = "teddy.bmp";  

//	Mat src = cv::imread(src_path, CV_LOAD_IMAGE_COLOR);   //srcΪԭͼ
	Mat src = cv::imread(src_path, 0);   //srcΪԭͼ

	cv::imshow("ԭʼͼ��", src);

	int rows = src.rows;              //ԭʼͼ��ĸ߶�rows
	int cols = src.cols;              //ԭʼͼ��Ŀ��cols
	int channels = src.channels();    //ԭʼͼ���ͨ����channels
	int out_rows = src.rows*scale;    //�任��ͼ��߶�rows
	int out_cols = src.cols*scale;    //�任��ͼ����cols

/*-------------------------------CPUͼ����-----------------------------*/
	Mat out(out_rows, out_cols, CV_8UC1);  //Ҫ�����ͼ��

	uchar *src_cpu_data = src.ptr<uchar>(0);   //ָ����src��һ�е�һ��Ԫ��
	uchar *out_cpu_data = out.ptr<uchar>(0);   //ָ����out��һ�е�һ��Ԫ��
	
	LARGE_INTEGER cpu_t1, cpu_t2, cpu_tc;
	QueryPerformanceFrequency(&cpu_tc);
	QueryPerformanceCounter(&cpu_t1);
	
	for (int time = 0; time < 100; time++)         //����100�Σ�ȡƽ��ֵ
	{
//		NearestInterpolation(src_cpu_data, out_cpu_data, scale,rows,cols,channels,out_rows,out_cols);
	}
	
	QueryPerformanceCounter(&cpu_t2);
	std::cout << "ʹ��CPU�����ڽ���ֵ����ʱ�䣺" << (cpu_t2.QuadPart - cpu_t1.QuadPart) * 1.0 * 1000 / cpu_tc.QuadPart /100 << "ms" << endl;



/*----------------------GPUͼ����-------------------------*/
	Mat gpu_out(out_rows, out_cols, CV_8UC3);
	uchar *src_gpu_data = src.ptr<uchar>(0);       //ָ����������ԭʼͼ��
	uchar *out_gpu_data = gpu_out.ptr<uchar>(0);   //ָ�������ɵ�GPU��

	const int nthreads = 256;
	dim3 bdim(nthreads, 1);
	dim3 gdim(divUp(out.cols, bdim.x), divUp(out.rows, bdim.y));

	LARGE_INTEGER gpu_t1, gpu_t2, gpu_tc;
	QueryPerformanceFrequency(&gpu_tc);
	QueryPerformanceCounter(&gpu_t1);
//	for (int time = 0; time < 100; time++)
//	{
		NI_kernel<nthreads> << <gdim, bdim >> > (src_gpu_data, out_gpu_data, scale, rows, cols, channels, out_rows, out_cols);
		hipDeviceSynchronize();
//	}
	QueryPerformanceCounter(&gpu_t2);
	cout << "ʹ��GPU�����ڽ���ֵ����ʱ�䣺" << (gpu_t2.QuadPart - gpu_t1.QuadPart) * 1.0 * 1000 / gpu_tc.QuadPart / 100 << "ms" << endl;




	cv::imshow("GPU�����ͼ��", gpu_out);//GPU�Ľ���������
	cv::imshow("CPU�����ͼ��", out);

	cv::waitKey(0);
	return 0;
}